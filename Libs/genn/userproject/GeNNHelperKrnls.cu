
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#define BlkSz 256 //NOTE: This was the best value for our machine even though it supports up to 1204 threads per block. You may try to change it to another value if it works better.

/*********************************/       
//kernels for random number generation
/*********************************/
__global__ void setup_kernel(hiprandState *state, unsigned long seed, int sizeofResult){
	int id = threadIdx.x+blockIdx.x*BlkSz;	
	if (id < sizeofResult) hiprand_init(seed, id, 0, &state[id]);
}

/*********************************/
template <class T>
__global__ void generate_random_gpuInput_xorwow(hiprandState * state, T * result, int sizeofResult, T Rstrength, T Rshift)
{
	int id = threadIdx.x+blockIdx.x* BlkSz; //TODO: use neuron kernel params
	T x;
	
	if (id < sizeofResult){
		hiprandState localstate = state[id];
		x =hiprand_normal(&localstate); 
		result[id] = x*Rstrength+Rshift;
		state[id]=localstate;
	}
}

/*********************************/
//function to setup the random number generator using the xorwow algorithm
/*********************************/
void xorwow_setup(hiprandState * devStates, long int sampleSize){
    int sampleBlkNo = ceilf(float(sampleSize/float(BlkSz)));
    dim3 sThreads(BlkSz,1);
    dim3 sGrid(sampleBlkNo,1); 

		long long int seed = 117; 
		setup_kernel<<<sGrid,sThreads>>>(devStates, seed, sampleSize);
}
