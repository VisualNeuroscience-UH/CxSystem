#include "hip/hip_runtime.h"

#ifndef TESTPREVARSINSYNAPSEDYNAMICS_CU
#define TESTPREVARSINSYNAPSEDYNAMICS_CU

#include <cstdlib>
#include <cfloat>
#include <iostream>
#include <fstream>

using namespace std;

#include "hr_time.cpp"
#include "utils.h"
#include "testHelper.h"

#include "testPostVarsInSynapseDynamics.h"
#include "postVarsInSynapseDynamics_CODE/definitions.h"
#include "postVarsInSynapseDynamics_CODE/runner.cc"



postVarsInSynapseDynamics::postVarsInSynapseDynamics()
{
  allocateMem();
  initialize();
  init_synapses();
  init_neurons();
}

postVarsInSynapseDynamics::~postVarsInSynapseDynamics()
{
  freeMem();
  delete[] theW;
}

void postVarsInSynapseDynamics::init_synapses() {
    theW= new float*[10];
    theW[0]= wsyn0;
    theW[1]= wsyn1;
    theW[2]= wsyn2;
    theW[3]= wsyn3;
    theW[4]= wsyn4;
    theW[5]= wsyn5;
    theW[6]= wsyn6;
    theW[7]= wsyn7;
    theW[8]= wsyn8;
    theW[9]= wsyn9;
}
    
void postVarsInSynapseDynamics::init_neurons() {
    for (int i= 0; i < 10; i++) {
	shiftpre[i]= i*10.0f;
	shiftpost[i]= i*10.0f;
    }
    copyStateToDevice();
}

void postVarsInSynapseDynamics::run(int which)
{
  if (which == GPU)
  {
    stepTimeGPU();
    copyStateFromDevice();
  }
  else
  {
    stepTimeCPU();
  }
}


/*====================================================================
--------------------------- MAIN FUNCTION ----------------------------
====================================================================*/

int main(int argc, char *argv[])
{
  if (argc != 4)
  {
    cerr << "usage: postVarsInSynapseDynamicsSim <GPU = 1, CPU = 0> <output label> <write output files? 0/1>" << endl;
    return EXIT_FAILURE;
  }

  postVarsInSynapseDynamics *sim = new postVarsInSynapseDynamics();
  int which= atoi(argv[1]);
  int write= atoi(argv[3]);
  CStopWatch *timer = new CStopWatch();
  string outLabel = toString(argv[2]);
  ofstream timeOs;
  ofstream neurOs;
  ofstream synOs;
  ofstream expSynOs;
  if (write) {
      timeOs.open((outLabel + "_time.dat").c_str(), ios::app);
      neurOs.open((outLabel + "_neur.dat").c_str());
      synOs.open((outLabel + "_syn.dat").c_str());
      expSynOs.open((outLabel + "_expSyn.dat").c_str());
  }
  float x[10][100];
  if (write) {
      cout << "# DT " << DT << endl;
      cout << "# TOTAL_TIME " << TOTAL_TIME << endl;
      cout << "# REPORT_TIME " << REPORT_TIME << endl;
      cout << "# begin simulating on " << ((which) ? "GPU" : "CPU") << endl;
  }
  timer->startTimer();
  float err= 0.0f;
  for (int d= 0; d < 10; d++) {
      for (int j= 0; j < 10; j++) {
	  for (int k= 0; k < 10; k++) {
	      x[d][j*10+k]= 0.0f;
	  }
      }
  }
  for (int i = 0; i < (TOTAL_TIME / DT); i++)
  {      
      t = i*DT;
      if (write) {
	  neurOs << t << " ";
	  synOs << t << " ";
	  expSynOs << t << " ";
      }
      for (int d= 0; d < 10; d++) { // for each delay
	  for (int j= 0; j < 10; j++) { // for all pre-synaptic neurons 
	      for (int k= 0; k < 10; k++) { // for all post-syn neurons
              // generate expected values
		  if (t > 0.0001+DT) {
		      x[d][j*10+k]= t-2*DT+10*k;
		  }

		  if (write) {
		      synOs << sim->theW[d][j*10+k] << " ";
		      expSynOs << x[d][j*10+k] << " ";
		  }
	      }
	  }		  
	  err+= absDiff(x[d], sim->theW[d], 100);
	  if (write) {
	      synOs << "    ";
	      expSynOs << "    ";
	  }
      }
      if (write) {
	  for (int j= 0; j < 10; j++) {
	      neurOs << xpost[j] << " ";
	  }
	  neurOs << "    ";
      }
      neurOs << endl;
      synOs << endl;
      expSynOs << endl;
      sim->run(which);
      if (fmod(t+5e-5, REPORT_TIME) < 1e-4)
      {
	  cout << "\r" << t;
      }
  }
  cout << "\r";
  timer->stopTimer();
  cout << "# done in " << timer->getElapsedTime() << " seconds" << endl;
  if (write) {
      timeOs << timer->getElapsedTime() << endl;
      timeOs.close();
      neurOs.close();
      synOs.close();
      expSynOs.close();
  }

  delete sim;
  delete timer;
  
  float tolerance= 5e-2;
  int success;
  string result;
  if (abs(err) < tolerance) {
      success= 1;
      result= tS("\033[1;32m PASS \033[0m");
  } else {
      success= 0;
      result= tS("\033[1;31m FAIL \033[0m");
  }
  cout << "# test postVarsInSynapseDynamics: Result " << result << endl;
  cout << "# the error was: " << err << " against tolerance " << tolerance << endl;
  cout << "#-----------------------------------------------------------" << endl;
  if (success)
      return EXIT_SUCCESS;
  else 
      return EXIT_FAILURE;
}

#endif // TESTPREVARSINSYNAPSEDYNAMICS_CU
